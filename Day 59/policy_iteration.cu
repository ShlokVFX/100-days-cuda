#include <stdio.h>
#include <hip/hip_runtime.h>

#define STATES 6
#define ACTIONS 2
#define GAMMA 0.9
#define THETA 1e-6

__global__ void evaluatePolicy(float *V, int *policy, float *rewards, float *transitions, bool *stable) {
    int s = threadIdx.x;
    float v_old = V[s];
    
    int a = policy[s];
    V[s] = 0.0;
    for (int next_s = 0; next_s < STATES; next_s++) {
        V[s] += transitions[s * STATES * ACTIONS + a * STATES + next_s] * (rewards[s * ACTIONS + a] + GAMMA * V[next_s]);
    }
    
    if (fabs(v_old - V[s]) > THETA) *stable = false;
}

__global__ void improvePolicy(float *V, int *policy, float *rewards, float *transitions, bool *stable) {
    int s = threadIdx.x;
    float best_value = -1e9;
    int best_action = 0;
    
    for (int a = 0; a < ACTIONS; a++) {
        float q_sa = 0.0;
        for (int next_s = 0; next_s < STATES; next_s++) {
            q_sa += transitions[s * STATES * ACTIONS + a * STATES + next_s] * (rewards[s * ACTIONS + a] + GAMMA * V[next_s]);
        }
        if (q_sa > best_value) {
            best_value = q_sa;
            best_action = a;
        }
    }
    
    if (policy[s] != best_action) {
        policy[s] = best_action;
        *stable = false;
    }
}

void policyIteration() {
    float V[STATES] = {0};
    int policy[STATES] = {0};
    float rewards[STATES * ACTIONS] = {0, 1, 0, 2, 0, 3, 0, 4, 1, 5, 0, 6};
    float transitions[STATES * STATES * ACTIONS] = {0.6, 0.4, 0, 0, 0, 0,  0.5, 0.5, 0, 0, 0, 0,
                                                     0, 0, 0.7, 0.3, 0, 0,  0, 0, 0.6, 0.4, 0, 0,
                                                     0, 0, 0, 0, 0.8, 0.2,  0, 0, 0, 0, 0.5, 0.5};
    
    float *d_V, *d_rewards, *d_transitions;
    int *d_policy;
    bool *d_stable, stable;
    hipMalloc(&d_V, STATES * sizeof(float));
    hipMalloc(&d_policy, STATES * sizeof(int));
    hipMalloc(&d_rewards, STATES * ACTIONS * sizeof(float));
    hipMalloc(&d_transitions, STATES * STATES * ACTIONS * sizeof(float));
    hipMalloc(&d_stable, sizeof(bool));
    
    hipMemcpy(d_V, V, STATES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_policy, policy, STATES * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rewards, rewards, STATES * ACTIONS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_transitions, transitions, STATES * STATES * ACTIONS * sizeof(float), hipMemcpyHostToDevice);
    
    int iteration = 0;
    do {
        stable = true;
        hipMemcpy(d_stable, &stable, sizeof(bool), hipMemcpyHostToDevice);
        evaluatePolicy<<<1, STATES>>>(d_V, d_policy, d_rewards, d_transitions, d_stable);
        improvePolicy<<<1, STATES>>>(d_V, d_policy, d_rewards, d_transitions, d_stable);
        hipMemcpy(&stable, d_stable, sizeof(bool), hipMemcpyDeviceToHost);
        
        hipMemcpy(V, d_V, STATES * sizeof(float), hipMemcpyDeviceToHost);
        printf("Iteration %d - V-values: ", iteration++);
        for (int s = 0; s < STATES; s++) printf("%.4f ", V[s]);
        printf("\n");
    } while (!stable);
    
    hipMemcpy(policy, d_policy, STATES * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Optimal Policy:\n");
    for (int s = 0; s < STATES; s++) printf("%d ", policy[s]);
    printf("\n");
    
    hipFree(d_V);
    hipFree(d_policy);
    hipFree(d_rewards);
    hipFree(d_transitions);
    hipFree(d_stable);
}

int main() {
    policyIteration();
    return 0;
}
