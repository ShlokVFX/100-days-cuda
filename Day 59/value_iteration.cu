#include <stdio.h>
#include <hip/hip_runtime.h>

#define STATES 6
#define ACTIONS 2
#define GAMMA 0.9
#define THETA 1e-6

__global__ void valueIterationStep(float *V, float *rewards, float *transitions, bool *stable) {
    int s = threadIdx.x;
    float v_old = V[s];
    float best_value = -1e9;
    
    for (int a = 0; a < ACTIONS; a++) {
        float q_sa = 0.0;
        for (int next_s = 0; next_s < STATES; next_s++) {
            q_sa += transitions[s * STATES * ACTIONS + a * STATES + next_s] * (rewards[s * ACTIONS + a] + GAMMA * V[next_s]);
        }
        if (q_sa > best_value) {
            best_value = q_sa;
        }
    }
    V[s] = best_value;
    
    if (fabs(v_old - V[s]) > THETA) *stable = false;
}

void valueIteration() {
    float V[STATES] = {0};
    float rewards[STATES * ACTIONS] = {0, 1, 0, 2, 0, 3, 0, 4, 1, 5, 0, 6};
    float transitions[STATES * STATES * ACTIONS] = {0.6, 0.4, 0, 0, 0, 0,  0.5, 0.5, 0, 0, 0, 0,
                                                     0, 0, 0.7, 0.3, 0, 0,  0, 0, 0.6, 0.4, 0, 0,
                                                     0, 0, 0, 0, 0.8, 0.2,  0, 0, 0, 0, 0.5, 0.5};
    
    float *d_V, *d_rewards, *d_transitions;
    bool *d_stable, stable;
    hipMalloc(&d_V, STATES * sizeof(float));
    hipMalloc(&d_rewards, STATES * ACTIONS * sizeof(float));
    hipMalloc(&d_transitions, STATES * STATES * ACTIONS * sizeof(float));
    hipMalloc(&d_stable, sizeof(bool));
    
    hipMemcpy(d_V, V, STATES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rewards, rewards, STATES * ACTIONS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_transitions, transitions, STATES * STATES * ACTIONS * sizeof(float), hipMemcpyHostToDevice);
    
    int iteration = 0;
    do {
        stable = true;
        hipMemcpy(d_stable, &stable, sizeof(bool), hipMemcpyHostToDevice);
        valueIterationStep<<<1, STATES>>>(d_V, d_rewards, d_transitions, d_stable);
        hipMemcpy(&stable, d_stable, sizeof(bool), hipMemcpyDeviceToHost);
        
        hipMemcpy(V, d_V, STATES * sizeof(float), hipMemcpyDeviceToHost);
        printf("Iteration %d - V-values: ", iteration++);
        for (int s = 0; s < STATES; s++) printf("%.4f ", V[s]);
        printf("\n");
    } while (!stable);
    
    printf("Optimal Value Function:\n");
    for (int s = 0; s < STATES; s++) printf("%.4f ", V[s]);
    printf("\n");
    
    hipFree(d_V);
    hipFree(d_rewards);
    hipFree(d_transitions);
    hipFree(d_stable);
}

int main() {
    valueIteration();
    return 0;
}
