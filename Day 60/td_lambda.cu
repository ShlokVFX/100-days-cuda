#include <iostream>
#include <hip/hip_runtime.h>

#define STATES 6
#define EPISODES 1000
#define ALPHA 0.1    // Learning rate
#define GAMMA 0.9    // Discount factor
#define LAMBDA 0.8   // Trace decay rate

__global__ void td_lambda_kernel(float *V, float *E, int *state_transitions, float *rewards) {
    int tid = threadIdx.x;

    if (tid < STATES) {
        float delta;
        for (int episode = 0; episode < EPISODES; episode++) {
            for (int t = 0; t < STATES - 1; t++) {
                int s = t;
                int s_next = state_transitions[t];

                // TD Error
                delta = rewards[s] + GAMMA * V[s_next] - V[s];

                // Update eligibility traces
                E[s] = E[s] * LAMBDA * GAMMA + 1.0f;

                // Update value function
                V[s] += ALPHA * delta * E[s];
            }
        }
    }
}

void td_lambda() {
    float V[STATES] = {0};  // State-value function
    float E[STATES] = {0};  // Eligibility traces
    int state_transitions[STATES] = {1, 2, 3, 4, 5, 5};  // State transitions
    float rewards[STATES] = {0, 0, 0, 0, 1, 0}; // Reward structure

    // Allocate device memory
    float *d_V, *d_E, *d_rewards;
    int *d_state_transitions;
    hipMalloc((void**)&d_V, STATES * sizeof(float));
    hipMalloc((void**)&d_E, STATES * sizeof(float));
    hipMalloc((void**)&d_state_transitions, STATES * sizeof(int));
    hipMalloc((void**)&d_rewards, STATES * sizeof(float));

    // Copy data to device
    hipMemcpy(d_V, V, STATES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_E, E, STATES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_state_transitions, state_transitions, STATES * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rewards, rewards, STATES * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    td_lambda_kernel<<<1, STATES>>>(d_V, d_E, d_state_transitions, d_rewards);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(V, d_V, STATES * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "TD(λ) Value Function:\n";
    for (int i = 0; i < STATES; i++) {
        std::cout << V[i] << " ";
    }
    std::cout << std::endl;

    // Free memory
    hipFree(d_V);
    hipFree(d_E);
    hipFree(d_state_transitions);
    hipFree(d_rewards);
}

int main() {
    td_lambda();
    return 0;
}
