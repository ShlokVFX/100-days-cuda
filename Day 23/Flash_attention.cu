
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void attention(float *Q, float *K, float *V, float *output, int n) {
    __shared__ float scores[4];
    __shared__ float softmax_scores[4];

    int tid = threadIdx.x;

    // Compute dot product (Q · K^T)
    scores[tid] = Q[tid] * K[tid];
    __syncthreads();

    // Print raw attention scores
    printf("Thread %d: Raw Score = %.2f\n", tid, scores[tid]);

    // Compute softmax denominator (sum of exponentials)
    float sum_exp = 0;
    for (int i = 0; i < n; i++) {
        sum_exp += expf(scores[i]);
    }

    // Compute softmax score
    softmax_scores[tid] = expf(scores[tid]) / sum_exp;
    __syncthreads();

    // Print softmax scores
    printf("Thread %d: Softmax Score = %.2f\n", tid, softmax_scores[tid]);

    // Compute weighted sum (Softmax × V)
    output[tid] = softmax_scores[tid] * V[tid];

    // Print final output
    printf("Thread %d: Output = %.2f\n", tid, output[tid]);
}

int main() {
    float h_Q[4] = {1, 0, 1, 0};
    float h_K[4] = {1, 1, 0, 0};
    float h_V[4] = {5, 10, 15, 20};
    float h_output[4] = {0};

    float *d_Q, *d_K, *d_V, *d_output;
    hipMalloc(&d_Q, 4 * sizeof(float));
    hipMalloc(&d_K, 4 * sizeof(float));
    hipMalloc(&d_V, 4 * sizeof(float));
    hipMalloc(&d_output, 4 * sizeof(float));

    hipMemcpy(d_Q, h_Q, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, 4 * sizeof(float), hipMemcpyHostToDevice);

    printf("Running Attention Kernel:\n");
    attention<<<1, 4>>>(d_Q, d_K, d_V, d_output, 4);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_output);

    return 0;
}
