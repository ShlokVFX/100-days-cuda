#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define N 3

int main() {
    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);

    float h_A[N * N] = { 4.0f, 12.0f, -16.0f,
                        12.0f, 37.0f, -43.0f,
                        -16.0f, -43.0f, 98.0f };

    printf("Input Matrix A (3x3):\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_A[i * N + j]);
        }
        printf("\n");
    }

    printf("\nFormula: A = L * L^T\n");
    printf("Extracting Lower Triangular Matrix L\n\n");

    float h_L[N * N];
    float *d_A;
    int *devInfo;
    int lwork;
    float *d_work;

    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&devInfo, sizeof(int));
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);

    hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, &lwork);
    hipMalloc((void**)&d_work, lwork * sizeof(float));

    hipsolverDnSpotrf(handle, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, d_work, lwork, devInfo);
    hipMemcpy(h_L, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Output Matrix L (Lower Triangular):\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (i >= j)
                printf("%f ", h_L[i * N + j]);
            else
                printf("0.000000 ");
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(handle);

    return 0;
}
