#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define N 3

int main() {
    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);

    float h_A[N * N] = { 3.0f, 1.0f, 0.0f,
                         1.0f, 2.0f, 0.0f,
                         0.0f, 0.0f, 1.0f };

    printf("Input Matrix A (3x3):\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_A[i * N + j]);
        }
        printf("\n");
    }

    printf("\nCalculating Eigenvalues & Eigenvectors using cuSolver\n\n");

    float h_W[N];  // Eigenvalues
    float *d_A, *d_W;
    int *devInfo;
    int lwork;
    float *d_work;

    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_W, N * sizeof(float));
    hipMalloc((void**)&devInfo, sizeof(int));
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);

    hipsolverDnSsyevd_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
                                N, d_A, N, d_W, &lwork);
    hipMalloc((void**)&d_work, lwork * sizeof(float));

    hipsolverDnSsyevd(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER,
                     N, d_A, N, d_W, d_work, lwork, devInfo);
    hipMemcpy(h_W, d_W, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_A, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Eigenvalues:\n");
    for (int i = 0; i < N; i++) {
        printf("%f\n", h_W[i]);
    }

    printf("\nEigenvectors (Column-wise):\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_A[i * N + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_W);
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(handle);

    return 0;
}
