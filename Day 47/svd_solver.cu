#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define M 3
#define N 2

int main() {
    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);

    float h_A[M * N] = { 3.0f, 1.0f,
                         1.0f, 3.0f,
                         0.0f, 1.0f };

    printf("Input Matrix A (3x2):\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_A[i * N + j]);
        }
        printf("\n");
    }

    printf("\nCalculating SVD: A = U * S * V^T\n\n");

    float h_S[N];          
    float h_U[M * M];      
    float h_VT[N * N];      
    float *d_A, *d_S, *d_U, *d_VT;
    int *devInfo;
    int lwork;
    float *d_work;
    float *rwork = NULL;

    hipMalloc((void**)&d_A, M * N * sizeof(float));
    hipMalloc((void**)&d_S, N * sizeof(float));
    hipMalloc((void**)&d_U, M * M * sizeof(float));
    hipMalloc((void**)&d_VT, N * N * sizeof(float));
    hipMalloc((void**)&devInfo, sizeof(int));

    hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);

    hipsolverDnSgesvd_bufferSize(handle, M, N, &lwork);
    hipMalloc((void**)&d_work, lwork * sizeof(float));

    char jobu = 'A'; 
    char jobvt = 'A'; 

    hipsolverDnSgesvd(handle, jobu, jobvt, M, N, d_A, M, d_S, d_U, M, d_VT, N, d_work, lwork, rwork, devInfo);

    hipMemcpy(h_S, d_S, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_U, d_U, M * M * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_VT, d_VT, N * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Singular Values (S):\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_S[i]);
    }
    printf("\n\n");

    printf("Left Singular Vectors (U):\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            printf("%f ", h_U[i * M + j]);
        }
        printf("\n");
    }

    printf("\nRight Singular Vectors (V^T):\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_VT[i * N + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_S);
    hipFree(d_U);
    hipFree(d_VT);
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(handle);

    return 0;
}
