#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define N_ARMS 10
#define N_TRIALS 1000
#define EPSILON 0.1

__global__ void bandit_kernel(float *estimates, int *counts, hiprandState *states, int n_trials) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState localState;
    
    if (tid == 0) {
        hiprand_init(1234, tid, 0, &localState);

        for (int t = 0; t < n_trials; t++) {
            float rand_val = hiprand_uniform(&localState);
            int action;
            if (rand_val < EPSILON) {
                action = hiprand(&localState) % N_ARMS;
            } else {
                action = 0;
                float max_estimate = estimates[0];
                for (int i = 1; i < N_ARMS; i++) {
                    if (estimates[i] > max_estimate) {
                        max_estimate = estimates[i];
                        action = i;
                    }
                }
            }

            float reward = hiprand_uniform(&localState);
            counts[action]++;
            estimates[action] += (reward - estimates[action]) / counts[action];
        }
    }
}

int main() {
    float *d_estimates, h_estimates[N_ARMS] = {0};
    int *d_counts, h_counts[N_ARMS] = {0};
    hiprandState *d_states;

    hipMalloc(&d_estimates, N_ARMS * sizeof(float));
    hipMalloc(&d_counts, N_ARMS * sizeof(int));
    hipMalloc(&d_states, sizeof(hiprandState));

    hipMemcpy(d_estimates, h_estimates, N_ARMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_counts, h_counts, N_ARMS * sizeof(int), hipMemcpyHostToDevice);

    bandit_kernel<<<1, 1>>>(d_estimates, d_counts, d_states, N_TRIALS);
    hipDeviceSynchronize();

    hipMemcpy(h_estimates, d_estimates, N_ARMS * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_counts, d_counts, N_ARMS * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Final Action Value Estimates:\n";
    for (int i = 0; i < N_ARMS; i++) {
        std::cout << "Arm " << i << ": " << h_estimates[i] << " (Chosen " << h_counts[i] << " times)\n";
    }

    hipFree(d_estimates);
    hipFree(d_counts);
    hipFree(d_states);

    return 0;
}
