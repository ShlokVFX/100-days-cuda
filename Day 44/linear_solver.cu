#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define N 3  // Size of the matrix

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        printf("%s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkCusolverError(hipsolverStatus_t err, const char* msg) {
    if (err != HIPSOLVER_STATUS_SUCCESS) {
        printf("%s\n", msg);
        exit(EXIT_FAILURE);
    }
}

int main() {

    float h_A[N * N] = { 1.0f, 2.0f, 3.0f,
                         4.0f, 5.0f, 6.0f,
                         7.0f, 8.0f, 10.0f };
    float h_B[N] = { 1.0f, 2.0f, 3.0f };
    
    float *d_A, *d_B;
    int *d_info, *d_pivot;
    
    checkCudaError(hipMalloc((void**)&d_A, N * N * sizeof(float)), "Failed to allocate d_A");
    checkCudaError(hipMalloc((void**)&d_B, N * sizeof(float)), "Failed to allocate d_B");
    checkCudaError(hipMalloc((void**)&d_info, sizeof(int)), "Failed to allocate d_info");
    checkCudaError(hipMalloc((void**)&d_pivot, N * sizeof(int)), "Failed to allocate d_pivot");

    checkCudaError(hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice), "Failed to copy A to device");
    checkCudaError(hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice), "Failed to copy B to device");


    hipsolverHandle_t handle;
    checkCusolverError(hipsolverDnCreate(&handle), "Failed to create cuSolver handle");

    // Workspace query
    int work_size = 0;
    checkCusolverError(hipsolverDnSgetrf_bufferSize(handle, N, N, d_A, N, &work_size),
                       "Failed to query buffer size");

    float *d_work;
    checkCudaError(hipMalloc((void**)&d_work, work_size * sizeof(float)), "Failed to allocate d_work");

    // LU Factorization
    checkCusolverError(hipsolverDnSgetrf(handle, N, N, d_A, N, d_work, d_pivot, d_info),
                       "Failed to perform LU factorization");

    // Solve AX = B
    checkCusolverError(hipsolverDnSgetrs(handle, HIPBLAS_OP_N, N, 1, d_A, N, d_pivot, d_B, N, d_info),
                       "Failed to solve linear system");

    checkCudaError(hipMemcpy(h_B, d_B, N * sizeof(float), hipMemcpyDeviceToHost),
                   "Failed to copy result to host");

    printf("Solution X:\n");
    for (int i = 0; i < N; i++) {
        printf("%f\n", h_B[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_work);
    hipFree(d_info);
    hipFree(d_pivot);
    hipsolverDnDestroy(handle);

    return 0;
}
