
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 10;
    int a[10], b[10], c[10];
    int *d_a, *d_b, *d_c;

    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    hipMalloc((void **)&d_a, n * sizeof(int));
    hipMalloc((void **)&d_b, n * sizeof(int));
    hipMalloc((void **)&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, n>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
