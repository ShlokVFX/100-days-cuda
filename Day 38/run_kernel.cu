#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA(call) \
    if((call) != hipSuccess) { \
        std::cerr << "CUDA Error at " << __LINE__ << std::endl; \
        return EXIT_FAILURE; \
    }

int main() {
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    CHECK_CUDA(hipInit(0));
    CHECK_CUDA(hipDeviceGet(&cuDevice, 0));
    CHECK_CUDA(hipCtxCreate(&cuContext, 0, cuDevice));
    CHECK_CUDA(hipModuleLoad(&cuModule, "cuda_kernel.ptx"));
    CHECK_CUDA(hipModuleGetFunction(&cuFunction, cuModule, "cuda_kernel"));

    // Allocate device memory
    float *d_data;
    hipMalloc(&d_data, 1024 * sizeof(float));
    hipMemset(d_data, 1.0, 1024 * sizeof(float));  // Initialize all to 1.0

    void *args[] = { &d_data };
    CHECK_CUDA(hipModuleLaunchKernel(cuFunction, 1, 1, 1, 1024, 1, 1, 0, 0, args, 0));

    // Copy back results
    float h_data[1024];
    hipMemcpy(h_data, d_data, 1024 * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result: " << h_data[0] << std::endl;  // Should print 2.0

    hipFree(d_data);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);
    return 0;
}
