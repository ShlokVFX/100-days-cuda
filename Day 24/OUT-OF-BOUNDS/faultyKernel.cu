#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void faultyKernel(int *d_arr, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Bug: No boundary check, leads to out-of-bounds access
    d_arr[idx] = idx * 2;  
}

 

int main() {
    int N = 100;  // Array size
    int *d_arr;

    // Allocate memory on the GPU
    hipMalloc((void **)&d_arr, N * sizeof(int));

    // Launching more threads than allocated memory
    int blockSize = 32;
    int numBlocks = 5;  // 5 * 32 = 160 threads, but only 100 elements in d_arr
    faultyKernel<<<numBlocks, blockSize>>>(d_arr, N);

    // Error checking
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    // Synchronize to catch errors
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error after sync: %s\n", hipGetErrorString(err));
    }

    // Free memory

    // Free memory
    hipFree(d_arr);
    return 0;
}
