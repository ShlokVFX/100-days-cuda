#include <iostream>
#include <hip/hip_runtime.h>

__global__ void reductionExample(int *d_out, int *d_in) {
    int val = d_in[threadIdx.x];

    // Use the built-in shuffle function
    for (int offset = 16; offset > 0; offset /= 2) {
        int temp = __shfl_down_sync(0xffffffff, val, offset);
        val += temp;
    }

    if (threadIdx.x == 0)
        d_out[0] = val;
}

int main() {
    const int N = 32;
    int h_in[N], h_out;
    int *d_in, *d_out;

    // Initialize with 1s
    for (int i = 0; i < N; i++) h_in[i] = 1 * 2;

    hipMalloc(&d_in, N * sizeof(int));
    hipMalloc(&d_out, sizeof(int));

    hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);

    reductionExample<<<1, N>>>(d_out, d_in);

    hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Warp Reduction Result: " << h_out << std::endl;

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
