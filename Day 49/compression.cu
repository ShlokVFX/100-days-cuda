#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <cstring>

#define PRUNE_THRESHOLD 0.0001f
#define N 4096

using namespace std;

__global__ void denseCompressionKernel(const float* input, float* output, float scale, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = roundf(input[idx] * scale) / scale;
    }
}

__global__ void pruneKernelOptimized(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < size; i += stride) {
        if (fabsf(data[i]) < PRUNE_THRESHOLD)
            data[i] = 0.0f;
    }
}

void generateRandomData(float* data, int size, float sparsity = 0.1f) {
    for (int i = 0; i < size; i++) {
        data[i] = (rand() % 10 < (sparsity * 10)) ? static_cast<float>(rand() % 100) : 0.0f;
    }
}

void compressSparseCSR(float* d_denseMatrix, int rows, int cols, hipsparseHandle_t handle, hipStream_t stream) {
    hipsparseSetStream(handle, stream);
    hipsparseDnMatDescr_t matA = nullptr;
    hipsparseSpMatDescr_t matB = nullptr;
    hipsparseCreateDnMat(&matA, rows, cols, rows, d_denseMatrix, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    int *d_csrRowPtr = nullptr, *d_csrColInd = nullptr;
    float *d_csrVal = nullptr;
    hipMalloc(&d_csrRowPtr, (rows + 1) * sizeof(int));
    hipMalloc(&d_csrColInd, rows * cols * sizeof(int));
    hipMalloc(&d_csrVal, rows * cols * sizeof(float));
    hipMemsetAsync(d_csrRowPtr, 0, (rows + 1) * sizeof(int), stream);
    hipsparseCreateCsr(&matB, rows, cols, 0, d_csrRowPtr, d_csrColInd, d_csrVal,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    pruneKernelOptimized<<<(rows * cols + 255) / 256, 256, 0, stream>>>(d_denseMatrix, rows * cols);
    hipStreamSynchronize(stream);
    size_t bufferSize = 0;
    void* dBuffer = nullptr;
    hipsparseDenseToSparse_bufferSize(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
    hipsparseDenseToSparse_analysis(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer);
    hipsparseDenseToSparse_convert(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer);
    int nnz;
    hipMemcpyAsync(&nnz, d_csrRowPtr + rows, sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    cout << "Sparse Compression (CSR) - Non-Zero Count: " << nnz << "\n";
    hipFree(dBuffer);
    hipsparseDestroyDnMat(matA);
    hipsparseDestroySpMat(matB);
    hipFree(d_csrRowPtr);
    hipFree(d_csrColInd);
    hipFree(d_csrVal);
}

int main() {
    srand(0);
    float* h_denseMatrix = nullptr;
    hipHostMalloc(&h_denseMatrix, N * N * sizeof(float), hipHostMallocDefault);
    generateRandomData(h_denseMatrix, N * N);
    float* d_denseMatrix = nullptr;
    hipMalloc(&d_denseMatrix, N * N * sizeof(float));
    hipMemcpy(d_denseMatrix, h_denseMatrix, N * N * sizeof(float), hipMemcpyHostToDevice);
    float* d_compressedDense = nullptr;
    hipMalloc(&d_compressedDense, N * N * sizeof(float));
    int initialNNZ = 0;
    for (int i = 0; i < N * N; i++) {
        if (h_denseMatrix[i] != 0.0f)
            initialNNZ++;
    }
    cout << "Initial Non-Zero Count: " << initialNNZ << "\n";
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipStream_t sparseStream;
    hipStreamCreate(&sparseStream);
    hipEventRecord(startEvent, 0);
    denseCompressionKernel<<<(N * N + 255) / 256, 256>>>(d_denseMatrix, d_compressedDense, 100.0f, N * N);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    float denseTime = 0.0f;
    hipEventElapsedTime(&denseTime, startEvent, stopEvent);
    cout << "Dense Compression Time: " << denseTime << " ms\n";
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, sparseStream);
    hipEventRecord(startEvent, sparseStream);
    compressSparseCSR(d_denseMatrix, N, N, handle, sparseStream);
    hipEventRecord(stopEvent, sparseStream);
    hipStreamSynchronize(sparseStream);
    float sparseTime = 0.0f;
    hipEventElapsedTime(&sparseTime, startEvent, stopEvent);
    cout << "Sparse Compression Time: " << sparseTime << " ms\n";
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipStreamDestroy(sparseStream);
    hipFree(d_denseMatrix);
    hipFree(d_compressedDense);
    hipHostFree(h_denseMatrix);
    hipsparseDestroy(handle);
    return 0;
}
