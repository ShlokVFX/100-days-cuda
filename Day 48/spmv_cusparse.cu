#include <iostream>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                                   \
    {                                                                      \
        hipError_t err = call;                                            \
        if (err != hipSuccess) {                                          \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << '\n'; \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

#define CHECK_CUSPARSE(call)                                              \
    {                                                                     \
        hipsparseStatus_t status = call;                                   \
        if (status != HIPSPARSE_STATUS_SUCCESS) {                          \
            std::cerr << "cuSPARSE error: " << status << '\n';            \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

int main() {
    const int A_num_rows = 4;
    const int A_num_cols = 4;
    const int A_nnz = 7;
    const int x_size = A_num_cols;
    const int y_size = A_num_rows;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    int hA_rowOffsets[] = {0, 1, 2, 5, 7};
    int hA_columns[] = {0, 1, 0, 1, 2, 0, 3};
    float hA_values[] = {0.5f, 1.0f, 0.2f, 0.3f, 1.2f, 0.4f, 1.1f};
    float hX[] = {1.0f, 1.0f, 1.0f, 1.0f};
    float hY[] = {0.0f, 0.0f, 0.0f, 0.0f};

    int *dA_rowOffsets, *dA_columns;
    float *dA_values, *dX, *dY;
    CHECK_CUDA(hipMalloc((void**)&dA_rowOffsets, (A_num_rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&dA_columns, A_nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&dA_values, A_nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dX, x_size * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dY, y_size * sizeof(float)));

    CHECK_CUDA(hipMemcpy(dA_rowOffsets, hA_rowOffsets, (A_num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dX, hX, x_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dY, hY, y_size * sizeof(float), hipMemcpyHostToDevice));

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                     dA_rowOffsets, dA_columns, dA_values,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, x_size, dX, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, y_size, dY, HIP_R_32F));

    size_t bufferSize = 0;
    void *dBuffer = nullptr;
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY,
         HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY,
         HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

    CHECK_CUDA(hipMemcpy(hY, dY, y_size * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Resultant vector y:\n";
    for (int i = 0; i < y_size; i++) {
        std::cout << hY[i] << " ";
    }
    std::cout << std::endl;

    CHECK_CUDA(hipFree(dA_values));
    CHECK_CUDA(hipFree(dA_columns));
    CHECK_CUDA(hipFree(dA_rowOffsets));
    CHECK_CUDA(hipFree(dX));
    CHECK_CUDA(hipFree(dY));
    CHECK_CUDA(hipFree(dBuffer));

    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));

    return 0;
}
