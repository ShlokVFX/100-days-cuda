#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipError_t res = hipInit(0);
    if (res == hipSuccess)
        std::cout << "CUDA Driver API initialized successfully!" << std::endl;
    else
        std::cout << "CUDA Driver API failed: " << res << std::endl;

    return 0;
}
