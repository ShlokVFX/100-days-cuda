#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA(call) \
    if((call) != hipSuccess) { \
        std::cerr << "CUDA Driver API error at " << __LINE__ << std::endl; \
        return EXIT_FAILURE; \
    }

int main() {
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    CHECK_CUDA(hipInit(0));
    CHECK_CUDA(hipDeviceGet(&cuDevice, 0));
    CHECK_CUDA(hipCtxCreate(&cuContext, 0, cuDevice));

    // Load the compiled CUBIN file
    CHECK_CUDA(hipModuleLoad(&cuModule, "vector_add.cubin"));
    CHECK_CUDA(hipModuleGetFunction(&cuFunction, cuModule, "vectorAdd"));

    // Allocate & initialize memory
    const int N = 1024;
    size_t size = N * sizeof(float);
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    hipDeviceptr_t d_A, d_B, d_C;
    CHECK_CUDA(hipMalloc(&d_A, size));
    CHECK_CUDA(hipMalloc(&d_B, size));
    CHECK_CUDA(hipMalloc(&d_C, size));

    CHECK_CUDA(hipMemcpyHtoD(d_A, h_A, size));
    CHECK_CUDA(hipMemcpyHtoD(d_B, h_B, size));

    // Launch kernel
    void* args[] = { &d_A, &d_B, &d_C, (void*)&N };
    CHECK_CUDA(hipModuleLaunchKernel(cuFunction,
                              (N + 255) / 256, 1, 1,  // Grid size
                              256, 1, 1,              // Block size
                              0, 0, args, 0));

    // Copy back result
    CHECK_CUDA(hipMemcpyDtoH(h_C, d_C, size));

    std::cout << "C[0] = " << h_C[0] << std::endl; // Expect 3.0

    // Cleanup
    delete[] h_A; delete[] h_B; delete[] h_C;
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    return 0;
}
