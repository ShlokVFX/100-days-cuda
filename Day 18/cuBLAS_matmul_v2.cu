#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>

#include <math.h>
#include <assert.h>
#include <cmath>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define EPSILON 1.0e-2

//verify result
void verify_result(float *a, float *b, float *c, int n) {
    float temp;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            temp = 0;
            for (int k = 0; k < n; k++) {
                temp += a[k * n + i] * b[j * n + k];  //column major
            }
            assert(fabs(c[j * n + i] - temp) < EPSILON);
        }
    }
}

int main() {
    //declare variables
    int n = 4; // Keeping small for readable output
    size_t bytes = n * n * sizeof(float);
    
    //declare pointers
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    //allocate memory
    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_c = (float *)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    //set seed
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    //generate random numbers
    hiprandGenerateUniform(prng, d_a, n * n);
    hiprandGenerateUniform(prng, d_b, n * n);

    //copy data back to host to print input matrices
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    
    printf("\nMatrix Size: %d x %d\n", n, n);
    
    printf("\nMatrix A:\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.4f ", h_a[i * n + j]);
        }
        printf("\n");
    }

    printf("\nMatrix B:\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.4f ", h_b[i * n + j]);
        }
        printf("\n");
    }

    //create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //scale factor
    float alpha = 1.0f;
    float beta = 0.0f;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //record start time
    hipEventRecord(start);

    //launch kernel
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

    //record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    double seconds = milliseconds / 1000.0;
    double gflops = (2.0 * pow(n, 3)) / (seconds * 1e9);

    //copy result back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    printf("\nMatrix C (Result):\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.4f ", h_c[i * n + j]);
        }
        printf("\n");
    }

    printf("\n🚀 Performance Metrics:\n");
    printf(" - Execution Time: %.6f ms\n", milliseconds);
    printf(" - GFLOPS: %.6f\n", gflops);

    //verify result
    verify_result(h_a, h_b, h_c, n);

    //free memory
    hipblasDestroy(handle);
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    printf("\nCOMPLETED SUCCESSFULLY\n");
    return 0;
}
