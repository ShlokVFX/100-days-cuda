#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>

#include <math.h>
#include <assert.h>
#include <cmath>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define EPSILON 1.0e-2

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, err); \
        exit(EXIT_FAILURE); \
    } \
}

//verify result
void verify_result(float *a, float *b, float *c, int n) {
    float temp;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            temp = 0;
            for (int k = 0; k < n; k++) {
                temp += a[k * n + i] * b[j * n + k];  //column major
            }
            assert(fabs(c[j * n + i] - temp) < EPSILON);
        }
    }
}

int main() {
    //declare variables
    int n = 1 << 10;
    size_t bytes = n * n * sizeof(float);
    
    //declare pointers
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    //allocate memory
    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_c = (float *)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    //set seed
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    //generate random numbers
    hiprandGenerateUniform(prng, d_a, n * n);
    hiprandGenerateUniform(prng, d_b, n * n);

    //create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    //scale factor
    float alpha = 1.0f;
    float beta = 0.0f;
    
    // Create and record CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //record start time
    hipEventRecord(start);

    //launch kernel
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n));

    //record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    double seconds = milliseconds / 1000.0;
    double gflops = (2.0 * n * n * n) / (seconds * 1e9);

    printf("\n🚀 Performance Metrics:\n");
    printf(" - Execution Time: %.6f ms\n", milliseconds);
    printf(" - GFLOPS: %.6f\n", gflops);

    //copy data back to host
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    //verify result
    verify_result(h_a, h_b, h_c, n);

    //free memory
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    printf("COMPLETED SUCCESSFULLY\n");

    return 0;
}
