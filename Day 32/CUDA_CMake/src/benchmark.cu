#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "vector_add.h"

inline void gpuCheck(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        std::cerr << "GPU Error: " << hipGetErrorString(code)
                  << " " << file << ":" << line << std::endl;
        exit(code);
    }
}
#define gpuErrchk(ans) { gpuCheck((ans), __FILE__, __LINE__); }

int main() {
    std::vector<size_t> testSizes = {1'000'000, 2'000'000, 10'000'000, 20'000'000, 50'000'000, 100'000'000, 500'000'000};

    std::cout << "  n (elements)  |  Runtime (ms)  |  GFLOPS\n"
              << "-----------------------------------------\n";

    for (auto n : testSizes) {
        std::vector<float> h_input1(n, 1.0f), h_input2(n, 2.0f), h_output(n, 0.0f);
        float *d_input1, *d_input2, *d_output;
        size_t bytes = n * sizeof(float);

        gpuErrchk(hipMalloc(&d_input1, bytes));
        gpuErrchk(hipMalloc(&d_input2, bytes));
        gpuErrchk(hipMalloc(&d_output, bytes));

        hipStream_t stream;
        hipStreamCreate(&stream);

        gpuErrchk(hipMemcpyAsync(d_input1, h_input1.data(), bytes, hipMemcpyHostToDevice, stream));
        gpuErrchk(hipMemcpyAsync(d_input2, h_input2.data(), bytes, hipMemcpyHostToDevice, stream));

        solution(d_input1, d_input2, d_output, n);

        gpuErrchk(hipMemcpyAsync(h_output.data(), d_output, bytes, hipMemcpyDeviceToHost, stream));
        hipStreamSynchronize(stream);
        hipStreamDestroy(stream);

        float milliseconds = 0.0f;
        hipEvent_t start, stop;
        gpuErrchk(hipEventCreate(&start));
        gpuErrchk(hipEventCreate(&stop));

        gpuErrchk(hipEventRecord(start));
        solution(d_input1, d_input2, d_output, n);
        gpuErrchk(hipEventRecord(stop));
        gpuErrchk(hipEventSynchronize(stop));
        gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));

        double gflops = (static_cast<double>(n) / (milliseconds / 1000.0)) / 1e9;
        std::cout << "  " << n << "           |  " << milliseconds << "          |  " << gflops << std::endl;

        gpuErrchk(hipFree(d_input1));
        gpuErrchk(hipFree(d_input2));
        gpuErrchk(hipFree(d_output));
    }

    return 0;
}
