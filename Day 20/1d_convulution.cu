#include <iostream>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

__global__ void conv1D(float *input, float *output, float *mask, int inputSize, int maskSize) {
    __shared__ float sharedInput[TILE_WIDTH + 2];

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = bx * TILE_WIDTH + tx;

    int maskRadius = maskSize / 2;

    if (index < inputSize) {
        sharedInput[tx + maskRadius] = input[index];
        if (tx < maskRadius) {
            if (index >= maskRadius) {
                sharedInput[tx] = input[index - maskRadius];
            } else {
                sharedInput[tx] = 0.0f;
            }
            if (index + TILE_WIDTH < inputSize) {
                sharedInput[tx + TILE_WIDTH + maskRadius] = input[index + TILE_WIDTH];
            } else {
                sharedInput[tx + TILE_WIDTH + maskRadius] = 0.0f;
            }
        }
    }

    __syncthreads();

    if (index < inputSize) {
        float result = 0.0f;
        for (int i = 0; i < maskSize; i++) {
            result += sharedInput[tx + i] * mask[i];
        }
        output[index] = result;
    }
}

void measureGflops(float *input, float *output, float *mask, int inputSize, int maskSize) {
    float *d_input, *d_output, *d_mask;
    hipMalloc(&d_input, inputSize * sizeof(float));
    hipMalloc(&d_output, inputSize * sizeof(float));
    hipMalloc(&d_mask, maskSize * sizeof(float));

    hipMemcpy(d_input, input, inputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, maskSize * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, 1, 1);
    dim3 dimGrid((inputSize + TILE_WIDTH - 1) / TILE_WIDTH, 1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    conv1D<<<dimGrid, dimBlock>>>(d_input, d_output, d_mask, inputSize, maskSize);
    hipEventRecord(stop);

    hipMemcpy(output, d_output, inputSize * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    
    printf("Time: %f ms\n", milliseconds);
    printf("GFLOPS: %f\n", 2.0 * inputSize * maskSize / (milliseconds * 1e6));
    printf("Output: %f\n", output[0]);
    printf("inputSize: %d\n", inputSize);
    printf("maskSize: %d\n", maskSize);
   // std::cout << "GFLOPS: " << gflops << std::endl;

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
}

int main() {
    int inputSize = 1024;
    int maskSize = 5;

    float *input = new float[inputSize];
    float *output = new float[inputSize];
    float *mask = new float[maskSize];

    for (int i = 0; i < inputSize; i++) {
        input[i] = static_cast<float>(i);
    }

    for (int i = 0; i < maskSize; i++) {
        mask[i] = static_cast<float>(i);
    }

    measureGflops(input, output, mask, inputSize, maskSize);

    delete[] input;
    delete[] output;
    delete[] mask;

    return 0;
}
