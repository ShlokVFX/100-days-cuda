#include <hip/hip_runtime.h>

#define TILE_SIZE 32
#define THREADS_PER_BLOCK 16

__global__ void matmul_kernel_optimized(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
                                       size_t M, size_t N, size_t K) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int blockRow = blockIdx.y * TILE_SIZE;
    int blockCol = blockIdx.x * TILE_SIZE;
    
    // Each thread computes 2x2 elements of C
    float sum00 = 0.0f, sum01 = 0.0f, sum10 = 0.0f, sum11 = 0.0f;
    
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        int tiledK = t * TILE_SIZE;
        
        // Load 2x2 tile from A into shared memory using float2
        int aRow = blockRow + ty * 2;
        int aCol = tiledK + tx * 2;
        if (aRow < M && aCol < K) {
            float2 aVal = *reinterpret_cast<const float2*>(&A[aRow * K + aCol]);
            As[ty * 2][tx * 2] = aVal.x;
            As[ty * 2][tx * 2 + 1] = aVal.y;
        } else {
            As[ty * 2][tx * 2] = 0.0f;
            As[ty * 2][tx * 2 + 1] = 0.0f;
        }
        if (aRow + 1 < M && aCol < K) {
            float2 aVal = *reinterpret_cast<const float2*>(&A[(aRow + 1) * K + aCol]);
            As[ty * 2 + 1][tx * 2] = aVal.x;
            As[ty * 2 + 1][tx * 2 + 1] = aVal.y;
        } else {
            As[ty * 2 + 1][tx * 2] = 0.0f;
            As[ty * 2 + 1][tx * 2 + 1] = 0.0f;
        }
        
        // Load 2x2 tile from B into shared memory using float2
        int bRow = tiledK + ty * 2;
        int bCol = blockCol + tx * 2;
        if (bRow < K && bCol < N) {
            float2 bVal = *reinterpret_cast<const float2*>(&B[bRow * N + bCol]);
            Bs[ty * 2][tx * 2] = bVal.x;
            Bs[ty * 2][tx * 2 + 1] = bVal.y;
        } else {
            Bs[ty * 2][tx * 2] = 0.0f;
            Bs[ty * 2][tx * 2 + 1] = 0.0f;
        }
        if (bRow + 1 < K && bCol < N) {
            float2 bVal = *reinterpret_cast<const float2*>(&B[(bRow + 1) * N + bCol]);
            Bs[ty * 2 + 1][tx * 2] = bVal.x;
            Bs[ty * 2 + 1][tx * 2 + 1] = bVal.y;
        } else {
            Bs[ty * 2 + 1][tx * 2] = 0.0f;
            Bs[ty * 2 + 1][tx * 2 + 1] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute 2x2 tile
        for (int i = 0; i < TILE_SIZE; ++i) {
            float a0 = As[ty * 2][i];
            float a1 = As[ty * 2 + 1][i];
            float b0 = Bs[i][tx * 2];
            float b1 = Bs[i][tx * 2 + 1];
            
            sum00 += a0 * b0;
            sum01 += a0 * b1;
            sum10 += a1 * b0;
            sum11 += a1 * b1;
        }
        __syncthreads();
    }
    
    // 2x2 result to global memory
    int cRow = blockRow + ty * 2;
    int cCol = blockCol + tx * 2;
    if (cRow < M && cCol < N) {
        C[cRow * N + cCol] = sum00;
        if (cCol + 1 < N) C[cRow * N + (cCol + 1)] = sum01;
        if (cRow + 1 < M) {
            C[(cRow + 1) * N + cCol] = sum10;
            if (cCol + 1 < N) C[(cRow + 1) * N + (cCol + 1)] = sum11;
        }
    }
}

extern "C" void solution(float* input_a, float* input_b, float* output_c,
                       size_t m, size_t n, size_t k) {
    dim3 block(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 grid((n + TILE_SIZE - 1) / TILE_SIZE,
              (m + TILE_SIZE - 1) / TILE_SIZE);
    
    matmul_kernel_optimized<<<grid, block>>>(input_a, input_b, output_c, m, n, k);
    hipDeviceSynchronize();
}