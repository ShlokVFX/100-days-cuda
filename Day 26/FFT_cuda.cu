#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <complex>

// Define complex number type
typedef std::complex<float> Complex;
#define IDX2C(i,j,ld) (((j)*(ld))+(i)) // Macro for column-major indexing

// CUDA Kernel to print device array (Debugging)
__global__ void printDeviceArray(hipfftComplex *data, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        printf("Device Data[%d]: (%f, %f)\n", i, data[i].x, data[i].y);
    }
}

// Host function to print host array
void printHostArray(const char* name, hipfftComplex *data, int N) {
    printf("\n%s:\n", name);
    for (int i = 0; i < N; i++) {
        printf("[%d]: (%f, %f)\n", i, data[i].x, data[i].y);
    }
}

// Main function
int main() {
    int N = 8; // Must be power of 2 for Cooley-Tukey FFT
    hipfftComplex *h_input, *h_output;
    hipfftComplex *d_data;

    // Allocate host memory
    h_input = (hipfftComplex*)malloc(sizeof(hipfftComplex) * N);
    h_output = (hipfftComplex*)malloc(sizeof(hipfftComplex) * N);

    // Initialize input data (Real and Imaginary parts)
    for (int i = 0; i < N; i++) {
        h_input[i].x = i + 1;  // Real part
        h_input[i].y = 0.0f;   // Imaginary part (set to 0)
    }
    
    // Print input data
    printHostArray("Input Data", h_input, N);

    // Allocate device memory
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * N);
    hipMemcpy(d_data, h_input, sizeof(hipfftComplex) * N, hipMemcpyHostToDevice);

    // Create FFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);

    // Execute FFT
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(h_output, d_data, sizeof(hipfftComplex) * N, hipMemcpyDeviceToHost);

    // Print output data
    printHostArray("Output Data (FFT Result)", h_output, N);

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_data);
    free(h_input);
    free(h_output);

    return 0;
}
